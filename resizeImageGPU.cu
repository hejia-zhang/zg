#include "hip/hip_runtime.h"
#include "resizeImageGPU.h"
#include ""

__global__ void kernel(unsigned char* d_res, const unsigned char* d_req, int n_resStep,
                       int n_reqStep, int n_resHeight, int n_resWidth, int n_reqHeight,
                       int n_reqWidth)
{
  int threadid = blockDim.x * blockIdx.x + threadIdx.x;

  int i = threadid / n_resWidth;
  int j = threadid % n_resWidth;
  
  double fRows = n_resHeight / (float)n_reqHeight;
  double fCols = n_resWidth / (float)n_reqWidth;

  int pX = 0;
  int pY = 0;

  pX = (int)(i / fRows);
  pY = (int)(j / fCols);
  if (pX < n_reqHeight && pX >= 0 && pY < n_reqWidth && pY >= 0)
  {
    *(d_res + i * n_resStep + 3 * j + 0) = *(d_req + pX * n_reqStep + 3 * pY + 0);
    *(d_res + i * n_resStep + 3 * j + 1) = *(d_req + pX * n_reqStep + 3 * pY + 1);
    *(d_res + i * n_resStep + 3 * j + 2) = *(d_req + pX * n_reqStep + 3 * pY + 2);
  }
}

void resizeImageGPU(unsigned char* d_res, const unsigned char* d_req, int n_resStep, 
                    int n_reqStep, int n_resHeight, int n_resWidth, int n_reqHeight, 
                    int n_reqWidth)
{
  int threadsPerBlock = 256;
  int blocksPerGrid = (n_resWidth * n_resHeight + threadsPerBlock - 1) / threadsPerBlock;
  kernel << < blocksPerGrid, threadsPerBlock>> > (d_res, d_req, n_resStep, n_reqStep,
                            n_resHeight, n_resWidth, n_reqHeight,
                            n_reqWidth);
}
